
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

__global__ void count_kernel(int N, const int *__restrict__ data,
                             int *__restrict__ out) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    if (data[i] >= 0)
      atomicAdd(&out[data[i]], 1);
  }
}

void count_wrapper(int N, const int *data, int *out) {
  count_kernel<<<ceil((double)N / 512), 512>>>(N, data, out);
}
